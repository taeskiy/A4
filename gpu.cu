#include "hip/hip_runtime.h"
/******************************************************************************
Hello! Ali here!!
  usage information and TODO stuff here...
Ali here again!!!
Hello Ali! Atai heree. Where are you?
******************************************************************************/


#include <stdio.h>
#include "gpu.h"

texture<float, 2> blueTex;

/******************************************************************************/
APopulation initializePop(unsigned int width, unsigned int height){

  APopulation P;
  P.nThreads.x = 32;  // 32 x 32 = 1024 threads per block
  P.nThreads.y = 32;
  P.nThreads.z = 1;
  P.nBlocks.x = (int) ceil(width/32.0);  // however many blocks needed for image
  P.nBlocks.y = (int) ceil(height/32.0);
  P.nBlocks.z = 1;
  P.pop_width = P.nBlocks.x * P.nThreads.x;       // save this info
  P.pop_height = P.nBlocks.y * P.nThreads.y;
  P.N = P.pop_width * P.pop_height;  // not the same as width and height

  hipError_t err;
  err = hipMalloc( (void**) &P.rand, P.N*sizeof(hiprandState));
  if(err != hipSuccess){
     printf("cuda error allocating rand = %s\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
     }

  err = hipMalloc( (void**) &P.red, P.N*sizeof(float));
  if(err != hipSuccess){
     printf("cuda error allocating red = %s\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
     }

  err = hipMalloc( (void**) &P.green, P.N*sizeof(float));
  if(err != hipSuccess){
     printf("cuda error allocating green = %s\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
     }

  err = hipMalloc( (void**) &P.blue, P.N*sizeof(float));
  if(err != hipSuccess){
     printf("cuda error allocating red = %s\n", hipGetErrorString(err));
     exit(EXIT_FAILURE);
     }


  setup_rands <<< P.nBlocks, P.nThreads >>> (P.rand, time(NULL), P.N);


  //----- placeholder for initializing memory with values
  // int a[P.N], b[P.N];
  // for (int i=0; i<P.N; i++){
  //   a[i] = -i;
  //   b[i] = i;
  // }
  // hipMemcpy(P.dev_a, a, P.N*sizeof(int), cH2D);
  // hipMemcpy(P.dev_b, b, P.N*sizeof(int), cH2D);
  // ------------------------

  return P;
}

/******************************************************************************/
int runIter(APopulation *P, unsigned long tick){

  printf("tick = %lu\n", tick);

  randomize <<< P->nBlocks, P->nThreads >>> (P->red, P->rand, P->N);
  randomize <<< P->nBlocks, P->nThreads >>> (P->green, P->rand, P->N);
  randomize <<< P->nBlocks, P->nThreads >>> (P->blue, P->rand, P->N);
  kernel <<< P->nBlocks, P->nThreads >>> (P->red, P->green, P->blue, P->N);


//  add <<< P->nBlocks, P->nThreads >>> (P->dev_a, P->dev_b, P->dev_c);


  // -- crud...
  // int a[P->N], b[P->N], c[P->N];
  // hipMemcpy(&a, P->dev_a, P->N * sizeof(int), cD2H);
  // hipMemcpy(&b, P->dev_b, P->N * sizeof(int), cD2H);
  // hipMemcpy(&c, P->dev_c, P->N * sizeof(int), cD2H);
  //
  // for(int i = 0; i< P->N; i++){
  //   printf("%d + %d = %d\n", a[i], b[i], c[i]);
  //   }
  // ----

  return 0;
}





/******************************************************************************/
// Mike Brady's Kernel
__global__ void
kernel(float* red, float* green, float* blue, unsigned long N){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  unsigned long tid = x + (y * blockDim.x * gridDim.x);

  if(tid < N){
      red[tid] = .5;
      blue[tid] = .7;
      green[tid]= .2;
    }
}

/******************************************************************************/
__global__ void
setup_rands(hiprandState* rand, unsigned long seed, unsigned long N)
{

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  unsigned long tid = x + (y * blockDim.x * gridDim.x);

  if(tid < N) hiprand_init(seed, tid, 0, &rand[tid]);

}

/******************************************************************************/
__global__ void
randomize(float* array, hiprandState* rand, unsigned long N)
{
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  unsigned long tid = x + (y * blockDim.x * gridDim.x);

  if(tid < N){
    hiprandState localState = rand[tid]; // get local hiprandState as seed
    float theRand = hiprand_uniform(&localState); // use to get value from 0-1
    rand[tid] = localState; // save new state as previous state for next gen

    array[tid] = theRand;
   }

}






/******************************************************************************/
void freeGPU(APopulation *P)
{
  hipFree(P->red);
  hipFree(P->green);
  hipFree(P->blue);
  hipFree(P->rand);

  //   hipFree(P->dev_a);
  // hipFree(P->dev_b);
  // hipFree(P->dev_c);
}

/******************************************************************************/
